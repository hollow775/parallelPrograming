#include "hip/hip_runtime.h"
#include "cuda.runtime.h"
#include ""

__global__ void GrayScale_CUDA(unsigned char *Image, int canais);

void Image_GrayScale_Cuda(unsigned char *Input_Image, int altura, int largura, int canais)
{
    unsigned char *Dev_Input_Image = NULL;
    hipMalloc((void **)&Dev_Input_Image, altura * largura * canais);
    hipMemcpy(Dev_Input_Image, Input_Image, altura * largura * canais, hipMemcpyHostToDevice);
    dim3 Grid_Image(largura, altura);
    GrayScale_CUDA<<<Grid_Image, 1>>>(Dev_Input_Image, canais);
    hipMemcpy(Input_Image, Dev_Input_Image, altura * largura * canais, hipMemcpyDeviceToHost);
    hipFree(Dev_Input_Image);
}

__global__ void GrayScale_CUDA(unsigned char *Image, int canais)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = (x + y * gridDim.x) * canais;

    float b = Imagem[idx + 0] float g = Imagem[idx + 1] float r = Imagem[idx + 2]

        for (int i = 0; i < canais; i++)
    {
        Imagem[idx + i] = b * 0.299f + g * 0.587f + r * 0.144f;
    }
}